﻿#include<iostream>
#include"hip/hip_runtime.h"

#include<cstdio>
#include<cmath>
using namespace std;

#define N 1024
#define B 256
#define O 256

float Rand() {
	return (2*float(rand())/RAND_MAX - 1)*0.1;
}

__device__ void convolucao(float* x, float* h, float* z) {
	float V;
	int j;
	for (int i = 0; i < N; i++) {
		V = 0;
		for (j = 0; j < N; j++) V += x[j]*h[i - j + (i >= j ? 0 : N)];
		z[i] = V;
	}
}

__global__ void convCompleta(float *x, float *y, float *z) {

    int L = (blockIdx.x*blockDim.x + threadIdx.x)*N*O;
	int i, j, k;
	x += L;
	y += L;
	z += L;

    for(int k = 0; k < O; k++) {
        convolucao(x, y, z);
        x += N;
        y += N;
        z += N;
    }
}

int main(void) {

	srand(10);
	setlocale(LC_ALL, "Portuguese");

	// Variáveis genéricas
	int i, j, k;
	float *V, *S, L;

	int BN = B*N;
	float *X = new float[BN];
	float *Y = new float[BN];
	float *Z = new float[BN];
	for(i = 0; i < BN; i++) {
        X[i] = Rand();
        Y[i] = Rand();
	}

	float *x, *y, *z;
	int d = sizeof(float);
	hipMalloc((void**)&x, d*BN);
	hipMalloc((void**)&y, d*BN);
	hipMalloc((void**)&z, d*BN);

	hipMemcpy(x, X, d*BN, hipMemcpyHostToDevice);
	hipMemcpy(y, Y, d*BN, hipMemcpyHostToDevice);

	cout << "RUN!!\n";
	float time;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	convCompleta<<<1, B/O>>>(x, y, z);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);

	hipMemcpy(Z, z, d*BN, hipMemcpyDeviceToHost);
	cout << "Time: " << time/1000 << " sec\n";


	getchar();
	for(i = 0; i < B; i++) {
		L = 0;
		V = Z + i*N;
		for (j = 0; j < N; j++) L += V[j];
		cout << i << ": " << L/N;

		if(i % 15 == 14) getchar();
        else cout << endl;
	}

	hipFree(x);
	hipFree(y);
	hipFree(z);

	return 0;
}
